#include "hip/hip_runtime.h"
#define WINDOW_WIDTH  1024
#define WINDOW_HEIGHT 1024

#include "cutil_math.h"

// Inifinity definition
#define CUDA_INF __int_as_float(0x7f800000)


// types of materials
#define MATERIAL_DIFFUSE 1  // A perfectly diffuse material
#define MATERIAL_SPECULAR 2 // A perfectly specular material

struct __align__(16) Sphere
{
    float3 center;
    float  radius;
    
    // Emission values, assume that the sphere emits diffusely at all points
    float3 emissionCol;

    int materialType;

    // Proportion of light reflected when light approaches the point along
    // the normal. For perfectly diffuse objects, this value represents the
    // reflectance along all directions.
    float3 reflectance;

};

struct __align__(16) Ray
{
    float3 o;
    float3 d;
};

struct __align__(16) EyePathNode
{
    float3 emission;
    float3 reflectance;
    int reflectOff; // Material type hit
};

// The maximum depth to go up to. This is hardcoded in compile time.
#define MAX_DEPTH 20

// Number of samples taken so far
__constant__ uint d_sampleNum;

__global__  void raytrace(unsigned char * out, int width, int height,
                          float vFov, Sphere * spheres, int numSpheres,
                          uint2 * seeds);


/////////////////////////////////////////////////////////////////////////////
// A simple multiply with carry 32 bit random number generator. The state
// of the random number generator is carried in the arguments. Taken
// from the Wikipedia page on random number generation.
/////////////////////////////////////////////////////////////////////////////
__device__ unsigned int getRand(uint2 * seed)
{
    seed->x = 36969 * (seed->x & 0xFFFF) + (seed->y >> 16);
    seed->y = 18000 * (seed->y & 0xFFFF) + (seed->x >> 16);

    return (seed->y << 16) + seed->x; 
}

/////////////////////////////////////////////////////////////////////////////
// Return a random float from 0 to 1
/////////////////////////////////////////////////////////////////////////////
__device__ float getRandFloat(uint2 * seed)
{
    return ((float)getRand(seed)) / 4294967296.0;
}

/////////////////////////////////////////////////////////////////////////////
// Function to calculate the intersection point of a ray with a sphere. 
// Assumes the ray has unit length
////////////////////////////////////////////////////////////////////////////
__device__ float rayIntersectSphere(Ray ray, float3 center, float radius)
{
    float3 oMinusC = ray.o - center; 
    float  dDotOC  = dot(ray.d, oMinusC);
    float disc = dDotOC * dDotOC - (dot(oMinusC, oMinusC) - radius * radius);

    if (disc < 0)
        return CUDA_INF;

    disc = sqrt(disc);

    if (-dDotOC - disc > 0)
        return -dDotOC - disc;

    if (-dDotOC + disc > 0)
        return -dDotOC + disc;

    return CUDA_INF;
}


///////////////////////////////////////////////////////////////////////////////
// Return the nearest sphere index. If none intesect, -1 is returned
///////////////////////////////////////////////////////////////////////////////
__device__ int nearestSphere(Ray r, Sphere * spheres, int numSpheres,
                             float3 * intersectP)
{
    int best = -1;
    float dist = CUDA_INF; 

    for (int i = 0; i < numSpheres; i++)
    {
        float thisDist = rayIntersectSphere(r, spheres[i].center,
                                            spheres[i].radius);
        if (thisDist < dist)
        {
            dist = thisDist;
            best = i;
            *intersectP = r.o + dist * r.d;
        }
    }

    return best;
}

///////////////////////////////////////////////////////////////////////////////
// Average in the current sample
///////////////////////////////////////////////////////////////////////////////
__device__ void addSample(unsigned char * out, int i, uint4 sample)
{
    
    out[4 * i]   = (out[4 * i]  * d_sampleNum + sample.x)   / (d_sampleNum + 1);
    out[4 * i+1] = (out[4 * i+1] * d_sampleNum + sample.y) / (d_sampleNum + 1);
    out[4 * i+2] = (out[4 * i+2] * d_sampleNum + sample.z) / (d_sampleNum + 1);
    out[4 * i+3] = (out[4 * i+3] * d_sampleNum + sample.w) / (d_sampleNum + 1);
    
}
///////////////////////////////////////////////////////////////////////////////
// Return a random reflection vector given a normal using cosine density
// (i.e. more likely to reflect toward the normal)
///////////////////////////////////////////////////////////////////////////////
__device__ float3 getRandRef(float3 normal, uint2 * seed)
{
    // Get a random rotation (phi) about the normal and angle off the plane
    // (theta) perpendicular to the normal
    float cosTheta = sqrt(1 - getRandFloat(seed));
    float sinTheta = sqrt(1 - cosTheta * cosTheta);
    float phi = 2.0 * 3.14 * getRandFloat(seed);

    // Construct some uvw basis respective to the normal, such that the
    // w axis goes along the normal.
    float3 w = normal;

    // initial value for v can be arbitrary as long as its not parallel
    // to the normal
    float3 v = normalize(normal + make_float3(normal.z, normal.x, normal.y));
    
    // calculate v and u using cross products, note that sign doesn't really
    // matter
    float3 u = cross(w, v);
    v = cross(w, u);

    // Use the basis and the angles to calculate the reflection
    float3 ref = u * (sinTheta * cos(phi))
               + v * (sinTheta * sin(phi))
               + w * (cosTheta);

    return normalize(ref);
}

///////////////////////////////////////////////////////////////////////////////
// Return the perfectly reflected vector
///////////////////////////////////////////////////////////////////////////////
__device__ float3 getRefPerfect(float3 n, float3 in)
{
    return in - 2.0 * dot(in, n) * n;
}

///////////////////////////////////////////////////////////////////////////////
// Return the sphere normal at some point
///////////////////////////////////////////////////////////////////////////////
__device__ float3 getNormal(Sphere s, float3 p)
{
    return normalize(p - s.center);
}

///////////////////////////////////////////////////////////////////////////////
// Build a path, collecting light samples along the way. The depth traveled
// is stored in depth.
///////////////////////////////////////////////////////////////////////////////
__device__ void buildPath(int i, Ray r, Sphere * spheres, int numSpheres,
                          uint2 * seed, int * depth, EyePathNode * pathInf)
{
    int dCount = 0; // How far the path is

    while (dCount < MAX_DEPTH)
    {
        // Get the next sphere to hit
        float3 p;
        int next = nearestSphere(r, spheres, numSpheres, &p);

        if (next == -1)
            break;

        // Store the emission of this sphere
        pathInf[dCount].emission = spheres[next].emissionCol; 

        // Check the type of material to see the next ray to shoot and the
        // reflectance value
        float3 nextDir;
        if (spheres[next].materialType == MATERIAL_DIFFUSE)
        {
            // Diffuse uses cosine density to reflect
            nextDir = getRandRef(getNormal(spheres[next], p), seed);
            
            r.o = p;
            r.d = nextDir;

            // Reflectance is constant regardless of direction
            pathInf[dCount].reflectance = spheres[next].reflectance;

            pathInf[dCount].reflectOff = MATERIAL_DIFFUSE;
        }
        else if (spheres[next].materialType == MATERIAL_SPECULAR)
        {
            // Perfectly specular surfaces perfect reflect the ray
            float3 n = getNormal(spheres[next], p);
            nextDir = getRefPerfect(n, r.d);
            
            r.o = p;
            r.d = nextDir;

            // Reflectance is modified depending on how the ray hits
            // the material

            // One minus cos (theta), theta is the angle the reflection
            // makes off the normal
            float omct = 1 - dot(nextDir, n);
            float omct5 = omct * omct * omct * omct * omct;
            
            pathInf[dCount].reflectance = spheres[next].reflectance
                                    + (make_float3(1.0, 1.0, 1.0)
                                       - spheres[next].reflectance) * omct5; 

            pathInf[dCount].reflectOff = MATERIAL_SPECULAR;
        }

        dCount ++;
    }

    *depth = dCount;
}

///////////////////////////////////////////////////////////////////////////////
// Combine the path information to get a sample
//////////////////////////////////////////////////////////////////////////////
__device__ float3 getSample(int i, int depth, EyePathNode * pathInf)
{
    float3 sample = pathInf[depth - 1].emission;
    depth--;
    while (depth > 0)
    {
        sample = pathInf[depth-1].emission
               + pathInf[depth-1].reflectance * sample;
        depth --;
    }

    return sample;
}

///////////////////////////////////////////////////////////////////////////////
// The kernel function to handle one pass of the raytracer
///////////////////////////////////////////////////////////////////////////////
__global__  void raytrace(unsigned char * out, int width, int height,
                          float vFov, Sphere * spheres, int numSpheres,
                          uint2 * seeds)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
   
    extern __shared__ Sphere s_spheres[];

    // Load up spheres into shared memory
    int startI = 0;
    while (startI < numSpheres)
    {
        if (startI + threadIdx.x < numSpheres)
           s_spheres[startI + threadIdx.x] = spheres[startI + threadIdx.x];

        startI += blockDim.x;
    }
  
    // Get the random seed for this pixel
    uint2 seed = seeds[i];

    // Scale x and y to a range to get the desired vertical field of view angle
    // and keep the aspect ratio.
    float maxY = 1.0 / tan(vFov/2);

    float x = ((float(i % width) * 2.0 / width) - 1.0) / maxY;
    float y = ((float(i / width) * 2.0 / height) - 1.0) / maxY;

    // Jiggle the ray a bit to get natural anti-aliasing
    x += (getRandFloat(&seed) * 2.0 - 1.0) / width / maxY;
    y += (getRandFloat(&seed) * 2.0 - 1.0) / height / maxY;

    // Generate the initial ray for this pixel
    Ray r;
    r.o = make_float3(0);
    r.d = normalize(make_float3(x, y, 1.0));

    // Recursively build up a path and store necessary light information
    EyePathNode pathInf[MAX_DEPTH];
    int depth;
    buildPath(i, r, s_spheres, numSpheres, &seed, &depth, pathInf); 

    float3 sample = getSample(i, depth, pathInf);
    addSample(out, i, make_uint4(min(sample.x * 255, 255.0),
                                 min(sample.y * 255, 255.0),
                                 min(sample.z * 255, 255.0),
                                 255));
    // Write the seed back
    seeds[i] = seed;
}
